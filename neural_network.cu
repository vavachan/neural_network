#include "hip/hip_runtime.h"
#include "neural_network.h"
#include <cmath>
#include <iostream>

layer::layer(int input_S, int output_S)
{
	inp_size = input_S;
	out_size = output_S;
	Weight  = new Matrix(inp_size,out_size);
	WeightT  = new Matrix(out_size,inp_size);
	dWeight = new Matrix(inp_size,out_size);
	Bias    = new Matrix(1,out_size);
	dBias   = new Matrix(1,out_size);


	/* for the time being we will initialize all the 
	   weigths and biases to zero, see what happens. */

	for(int i=0;i<Weight->dim_x;i++)  
	{
		for(int j=0;j<Weight->dim_y;j++)
		{
			Weight->M[j*Weight->dim_x+i] = 1.;//rand()%10;
			//B->M[j*sizeX+i] = rand()%10;
		}
	}
	for(int i=0;i<Bias->dim_x;i++)  
	{
		for(int j=0;j<Bias->dim_y;j++)
		{
			Bias->M[j*Bias->dim_x+i] = 2.;//rand()%10;
			//B.M[j*sizeX+i] = rand()%10;
		}
	}
}
void layer::forward(Matrix* input, Matrix* output)
{
	/* the dimension of the  input vector is input->dim_y ( basically a 1dim column vector)
	   the  dimension of the  out vector is correspondingly output->dim_y. 
   	   These vectors are given to the layer, the only job of the  layer is to 
	   compute output[i] = Weight[i][j]*input[j]+Bias[i] 
	   Therefore the Weight->dim_x has to match input->dim_y 
	   Such checks are  done here ? // why o why
	 */
	matrix_multiply_add_gpu(Weight,input,Bias,output);
}

void layer::backward(Matrix* gradient_of_this_layer,Matrix* gradient_from_prev_layer)
{
	/* This is backward propogation part of the layer. 
	   Here we find the derivative of the cost function 
	   with respect to the weights associated with this layer. 
	   This will have contribution due to the derivatives 
	   of all the previous layers. This is given to the function 
	   as delta. The only thing this function is supposed to do 
	   is multiply the transpose of weights of this layer with delta. 
	   And this is the new delta which will be passed on to the  
	   previous layer after multiplying the elements with 
	   the derivative of the activation function etc. To find the derivative of cost function 
	   with respect to the weights of this layer, we only 
	   need the delta and the input value from the previous layer.  
	 */
	
}
sigmoid_layer::sigmoid_layer(int input_S, int output_S) : layer(input_S, output_S)
{
	;//layer::layer(int input_S, int output_S);
}


void sigmoid_layer::forward(Matrix* input, Matrix* output)
{
	//layer::forward(input,output);
	sigmoid_layer_forward_gpu(input,output);

}

void sigmoid_layer_forward_gpu(Matrix* input, Matrix* output)
{
	int block_size = 32;

	if(!((input->dim_x == output->dim_x) and (input->dim_y == output->dim_y)))
	{
		std::cout<<"error dimension miss match \n"<<"\n";
	}

	int n_blocks_x=(input->dim_x+block_size-1)/block_size; 
	int n_blocks_y=(input->dim_y+block_size-1)/block_size; 
	//cout<<n_blocks_x<<"\t"<<n_blocks_y<<"\n";

	dim3 n_blocks(n_blocks_x,n_blocks_y);
	dim3 n_threads(block_size,block_size);
	//cout<<"time\t"<<time(0)<<"\n";

	sigmoid_function <<< n_blocks,n_threads >>> (input->M,output->M,input->dim_x,input->dim_y,output->dim_x,output->dim_y);
	hipDeviceSynchronize();
}

__global__
void sigmoid_function(float *input, float *output, int input_dim_x, int input_dim_y, int output_dim_x, int output_dim_y)
{
	int col = blockDim.x*blockIdx.x+threadIdx.x;
	int row = blockDim.y*blockIdx.y+threadIdx.y;
	if(col < output_dim_x and row < output_dim_y)
	{
		output[row*output_dim_x+col] = 1/(1+exp(input[row*input_dim_x+col]));
	}

}

void mean_squared_error_2d_gpu(Matrix* predictions, Matrix* target, Matrix *gradient, float *error)
{
	int block_size = 32;

	if(!((predictions->dim_x == target->dim_x) and (predictions->dim_y == target->dim_y)))
	{
		std::cout<<"error dimension miss match \n"<<"\n";
	}

	int n_blocks_x=(predictions->dim_x+block_size-1)/block_size; 
	int n_blocks_y=(predictions->dim_y+block_size-1)/block_size; 
	//cout<<n_blocks_x<<"\t"<<n_blocks_y<<"\n";

	dim3 n_blocks(n_blocks_x,n_blocks_y);
	dim3 n_threads(block_size,block_size);
	//cout<<"time\t"<<time(0)<<"\n";

	//std::cout<<*error<<"\t"<<predictions->dim_x<<"\t"<<predictions->dim_y<<"\n";
	mean_squared_error_2d <<< n_blocks,n_threads >>> (predictions->M,target->M,gradient->M,predictions->dim_x, predictions->dim_y, error);
	//float cost =0. ;
	//for(int i=0; i < predictions->dim_y; i++)
	//{
	//	cost = cost+(predictions->M[i]-target->M[i])*(predictions->M[i]-target->M[i]);
	//}
	//cost = cost/predictions->dim_y;
	////std::cout<<cost<<"\n";
	hipDeviceSynchronize();
}
__global__ 
void mean_squared_error_2d(float *predictions, float *target, float *gradient, int size_x, int size_y, float *error)
{
	int col = blockDim.x*blockIdx.x+threadIdx.x;
	int row = blockDim.y*blockIdx.y+threadIdx.y;
	//atomicAdd(error,1);
	if(col < size_x and row < size_y)
	{
		//output[row*output_dim_x+col] = 1/(1+exp(input[row*input_dim_x+col]));
		atomicAdd(error,fdividef(powf(predictions[row*size_x+col]-target[row*size_x+col],2),size_x*size_y));
		gradient[row*size_x+col] = fdividef(2.*(predictions[row*size_x+col]-target[row*size_x+col]),size_x*size_y) ;
	}

}

//
